#include "hip/hip_runtime.h"
///----------------------------------*-C++-*----------------------------------//
/*!
 * \file   device/test/Dual_Call.cu
 * \author Alex R. Long
 * \date   Mon Mar 25 2019
 * \brief  Show how code can be called from GPU and host
 * \note   Copyright (C) 2019 Triad National Security, LLC.
 *         All rights reserved.
 */
//---------------------------------------------------------------------------//

#include "Dual_Call.hh"

namespace rtt_device_test {

//---------------------------------------------------------------------------//
/*!
 * \brief Calculate the number of source particles for a range of cells
 *
 * \param[in] part_per_e energy for this source
 * \param[in] max_particles_pspc max partices per species
 * \param[in] cell_start starting cell index
 * \param[in] cell_end ending cell index
 * \param[in] e_field energy in a cell
 * \param[in] src_cell_bias bias in a cell
 * \param[in,out] n_field destination for particles in a cell
 * \param[out] return number of particles over this cell range
 */
__host__ __device__ unsigned long long sub_conserve_calc_num_src_particles(
    const double part_per_e, unsigned max_particles_pspc,
    const size_t cell_start, const size_t cell_end, const double *e_field,
    const double *src_cell_bias, int *n_field) {
  unsigned long long ntot = 0;

  ntot = 0;

  // sweep through cells and calculate number of particles per cell
  for (size_t cell = cell_start; cell < cell_end; cell++) {
    // if the cell has any energy try to put some particles in it
    if (e_field[cell] > 0.0) {
      // get estimate of number of particles per cell to nearest
      // integer per species, a cell-based bias can be added that simply
      // multiplies the expected number by a user defined bias; the
      // energy balance will still be correct because particles will
      // simply be subtracted from other cells to compensate
      const double d_num = e_field[cell] * part_per_e * src_cell_bias[cell];
      //Check(d_num > 0.0);
      // Check( d_num < static_cast<double>(max_particles_pspc) );

      // We are about to cast d_num back to int.  Ensure that the
      // conversion is valid.  If not, set the number of particles to
      // the ceiling value provided in Source.hh.
      if (d_num < static_cast<double>(max_particles_pspc - 1)) {
        n_field[cell] = static_cast<int>(d_num + 0.5);

        // try to get at least one particle per cell per species
        if (n_field[cell] == 0)
          n_field[cell] = 1;

      } else {
        n_field[cell] = max_particles_pspc;
      }

      // increment particle counter (uint64_t += int)
      ntot += n_field[cell];
    } else
      n_field[cell] = 0;
  }
  return ntot;
}

//---------------------------------------------------------------------------//
/*!
 * \brief Launch a kernel to calculate the number of source particles
 *
 * \param[in] part_per_e energy for this source
 * \param[in] max_particles_pspc max partices per species
 * \param[in] cont_size size of all fields
 * \param[in] e_field energy in a cell
 * \param[in] src_cell_bias bias in a cell
 * \param[in,out] n_field destination for particles in a cell
 * \param[in,out] ntot total particles per thread block
 */
__global__ void cuda_conserve_calc_num_src_particles(
    const double part_per_e, unsigned max_particles_pspc, int cont_size,
    const double *e_field, const double *src_cell_bias, int *n_field,
    unsigned long long *ntot) {

  __shared__ unsigned long long shared_data[512];
  size_t cell_start = threadIdx.x + blockIdx.x * blockDim.x;
  size_t cell_end = cell_start + 1;
  if (cell_start < cont_size) {
    shared_data[threadIdx.x] = sub_conserve_calc_num_src_particles(
        part_per_e, max_particles_pspc, cell_start, cell_end, e_field,
        src_cell_bias, n_field);
  } else
    shared_data[threadIdx.x] = 0;
  __syncthreads();
  for (unsigned int s = 1; s < blockDim.x; ++s) {
    if (s == threadIdx.x)
      shared_data[0] += shared_data[s];
    __syncthreads();
  }
  __syncthreads();
  ntot[blockIdx.x] = shared_data[0];
}

} // namespace rtt_device_test

//---------------------------------------------------------------------------//
// end of device/test/Dual_Call.cc
//---------------------------------------------------------------------------//
